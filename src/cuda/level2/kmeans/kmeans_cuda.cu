#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>
#include <omp.h>
#include <hip/hip_runtime.h>

#define THREADS_PER_DIM 16
#define BLOCKS_PER_DIM 16
#define THREADS_PER_BLOCK THREADS_PER_DIM*THREADS_PER_DIM

#include "cudacommon.h"
#include "ResultDatabase.h"
#include "OptionParser.h"
#include "kmeans_cuda_kernel.cu"

//#define BLOCK_DELTA_REDUCE
//#define BLOCK_CENTER_REDUCE

#define CPU_DELTA_REDUCE
#define CPU_CENTER_REDUCE

int setup(ResultDatabase &resultDB, OptionParser &op);

// GLOBAL!!!!!
/* sqrt(256) -- see references for this choice */
unsigned int num_threads_perdim = THREADS_PER_DIM;					
/* temporary */
unsigned int num_blocks_perdim = BLOCKS_PER_DIM;
/* number of threads */
unsigned int num_threads = num_threads_perdim*num_threads_perdim;	
/* number of blocks */
unsigned int num_blocks = num_blocks_perdim*num_blocks_perdim;		

/* _d denotes it resides on the device */
/* newly assignment membership */
int    *membership_new;												
/* inverted data array */
float  *feature_d;													
/* original (not inverted) data array */
float  *feature_flipped_d;											
/* membership on the device */
int    *membership_d;												
/* sum of points in a cluster (per block) */
float  *block_new_centers;											
/* cluster centers on the device */
float  *clusters_d;													
/* per block calculation of cluster centers */
float  *block_clusters_d;											
/* per block calculation of deltas */
int    *block_deltas_d;												

////////////////////////////////////////////////////////////////////////////////
void addBenchmarkSpecOptions(OptionParser &op) {
    op.addOption("maxClusters", OPT_INT, "5", "maximum number of clusters allowed");
    op.addOption("minClusters", OPT_INT, "5", "minimum number of clusters allowed");
    op.addOption("threshold", OPT_FLOAT, "0.001", "threshold value");
    op.addOption("loops", OPT_INT, "1", "iteration for each number of clusters");
    op.addOption("rmse", OPT_BOOL, "0", "calculate RMSE (default off)");
    op.addOption("outputCenters", OPT_BOOL, "0", "output cluster center coordinates (default off)");
}

void RunBenchmark(ResultDatabase &resultDB, OptionParser &op) {
    printf("Running KMeans\n");
    setup(resultDB, op);
}
////////////////////////////////////////////////////////////////////////////////

/* -------------- allocateMemory() ------------------- */
/* allocate device memory, calculate number of blocks and threads, and invert the data array */
void allocateMemory(int npoints, int nfeatures, int nclusters, float **features)
{	
	num_blocks = npoints / num_threads;
	if (npoints % num_threads > 0)		/* defeat truncation */
		num_blocks++;

	num_blocks_perdim = sqrt((double) num_blocks);
	while (num_blocks_perdim * num_blocks_perdim < num_blocks)	// defeat truncation (should run once)
		num_blocks_perdim++;

	num_blocks = num_blocks_perdim*num_blocks_perdim;

	/* allocate memory for memory_new[] and initialize to -1 (host) */
#ifdef UNIFIED_MEMORY
    CUDA_SAFE_CALL(hipMallocManaged(&membership_new, npoints * sizeof(int)));
#else
	membership_new = (int*) malloc(npoints * sizeof(int));
#endif
	for(int i=0;i<npoints;i++) {
		membership_new[i] = -1;
	}

	/* allocate memory for block_new_centers[] (host) */
#ifdef UNIFIED_MEMORY
    CUDA_SAFE_CALL(hipMallocManaged(&block_new_centers, nclusters * nfeatures * sizeof(float)));
#else
	block_new_centers = (float *) malloc(nclusters*nfeatures*sizeof(float));
#endif
	
	/* allocate memory for feature_flipped_d[][], feature_d[][] (device) */
    // TODO change unnecessary copy
#ifdef UNIFIED_MEMORY
	//CUDA_SAFE_CALL(hipMallocManaged((void**) &feature_flipped_d, npoints*nfeatures*sizeof(float)));
	//CUDA_SAFE_CALL(hipMemcpy(feature_flipped_d, features[0], npoints*nfeatures*sizeof(float), hipMemcpyHostToDevice));
    feature_flipped_d = features[0];
	CUDA_SAFE_CALL(hipMallocManaged((void**) &feature_d, npoints*nfeatures*sizeof(float)));
#else
	hipMalloc((void**) &feature_flipped_d, npoints*nfeatures*sizeof(float));
	hipMemcpy(feature_flipped_d, features[0], npoints*nfeatures*sizeof(float), hipMemcpyHostToDevice);
	hipMalloc((void**) &feature_d, npoints*nfeatures*sizeof(float));
#endif
		
	/* invert the data array (kernel execution) */	
	invert_mapping<<<num_blocks,num_threads>>>(feature_flipped_d,feature_d,npoints,nfeatures);
    //CHECK_CUDA_ERROR();
		
	/* allocate memory for membership_d[] and clusters_d[][] (device) */
#ifdef UNIFIED_MEMORY
	//CUDA_SAFE_CALL(hipMallocManaged((void**) &membership_d, npoints*sizeof(int)));
	//CUDA_SAFE_CALL(hipMallocManaged((void**) &clusters_d, nclusters*nfeatures*sizeof(float)));
#else
	hipMalloc((void**) &membership_d, npoints*sizeof(int));
	hipMalloc((void**) &clusters_d, nclusters*nfeatures*sizeof(float));
#endif

	
#ifdef BLOCK_DELTA_REDUCE
	// allocate array to hold the per block deltas on the gpu side
#ifdef UNIFIED_MEMORY
	CUDA_SAFE_CALL(cudaMallociManaged((void**) &block_deltas_d, num_blocks_perdim * num_blocks_perdim * sizeof(int)));
#else
	hipMalloc((void**) &block_deltas_d, num_blocks_perdim * num_blocks_perdim * sizeof(int));
#endif
	//hipMemcpy(block_delta_d, &delta_h, sizeof(int), hipMemcpyHostToDevice);
#endif

#ifdef BLOCK_CENTER_REDUCE
	// allocate memory and copy to card cluster  array in which to accumulate center points for the next iteration
#ifdef UNIFIED_MEMORY
    CUDA_SAFE_CALL(hipMallocManaged((void**) &block_clusters_d, 
            num_blocks_perdim * num_blocks_perdim * 
            nclusters * nfeatures * sizeof(float)));

#else
    hipMalloc((void**) &block_clusters_d, 
            num_blocks_perdim * num_blocks_perdim * 
            nclusters * nfeatures * sizeof(float));
#endif
	//hipMemcpy(new_clusters_d, new_centers[0], nclusters*nfeatures*sizeof(float), hipMemcpyHostToDevice);
#endif

}
/* -------------- allocateMemory() end ------------------- */

/* -------------- deallocateMemory() ------------------- */
/* free host and device memory */
void deallocateMemory()
{
#ifdef UNIFIED_MEMORY
    CUDA_SAFE_CALL(hipFree(membership_new));
    CUDA_SAFE_CALL(hipFree(block_new_centers));
#else
	free(membership_new);
	free(block_new_centers);
    hipFree(membership_d);
	hipFree(clusters_d);
#endif
	hipFree(feature_d);
	//hipFree(feature_flipped_d);
	//hipFree(membership_d);

#ifdef BLOCK_CENTER_REDUCE
    hipFree(block_clusters_d);
#endif
#ifdef BLOCK_DELTA_REDUCE
    hipFree(block_deltas_d);
#endif
}
/* -------------- deallocateMemory() end ------------------- */


/* ------------------- kmeansCuda() ------------------------ */    
int	// delta -- had problems when return value was of float type
kmeansCuda(float  **feature,				/* in: [npoints][nfeatures] */
           int      nfeatures,				/* number of attributes for each point */
           int      npoints,				/* number of data points */
           int      nclusters,				/* number of clusters */
           int     *membership,				/* which cluster the point belongs to */
		   float  **clusters,				/* coordinates of cluster centers */
		   int     *new_centers_len,		/* number of elements in each cluster */
           float  **new_centers,			/* sum of elements in each cluster */
           double &transferTime,
           double &kernelTime,
		   ResultDatabase &resultDB)
{
	int delta = 0;			/* if point has moved */
	int i,j;				/* counters */

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float elapsedTime;

    hipEventRecord(start, 0);
    /* copy membership (host to device) */
#ifdef UNIFIED_MEMORY
    //CUDA_SAFE_CALL(hipMemcpy(membership_d, membership_new, npoints*sizeof(int), hipMemcpyHostToDevice));
    membership_d = membership_new;
    //CUDA_SAFE_CALL(hipMemcpy(clusters_d, clusters[0], nclusters*nfeatures*sizeof(float), hipMemcpyHostToDevice));
    clusters_d = clusters[0];
#else
    hipMemcpy(membership_d, membership_new, npoints*sizeof(int), hipMemcpyHostToDevice);
    /* copy clusters (host to device) */
    hipMemcpy(clusters_d, clusters[0], nclusters*nfeatures*sizeof(float), hipMemcpyHostToDevice);
#endif
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    transferTime += elapsedTime * 1.e-3; // convert to seconds

    hipError_t err;

	/* set up texture */
    hipChannelFormatDesc chDesc0 = hipCreateChannelDesc<float>();
    t_features.filterMode = hipFilterModePoint;   
    t_features.normalized = false;
    t_features.channelDesc = chDesc0;

	err = hipBindTexture(NULL, &t_features, feature_d, &chDesc0, npoints*nfeatures*sizeof(float));
    if(err != hipSuccess) {
        printf("Error: Couldn't bind features array to texture, %d", err);
        exit(0);
    }

	hipChannelFormatDesc chDesc1 = hipCreateChannelDesc<float>();
    t_features_flipped.filterMode = hipFilterModePoint;   
    t_features_flipped.normalized = false;
    t_features_flipped.channelDesc = chDesc1;

	err = hipBindTexture(NULL, &t_features_flipped, feature_flipped_d, &chDesc1, npoints*nfeatures*sizeof(float));
    if(err != hipSuccess) {
        printf("Error: Couldn't bind features_flipped array to texture, %d", err);
        exit(0);
    }

	hipChannelFormatDesc chDesc2 = hipCreateChannelDesc<float>();
    t_clusters.filterMode = hipFilterModePoint;   
    t_clusters.normalized = false;
    t_clusters.channelDesc = chDesc2;

	err = hipBindTexture(NULL, &t_clusters, clusters_d, &chDesc2, nclusters*nfeatures*sizeof(float));
    if(err != hipSuccess) {
        printf("Error: Couldn't bind clusters array to texture, %d", err);
        exit(0);
    }

  hipEventRecord(start, 0);
	/* copy clusters to constant memory */
	hipMemcpyToSymbol(HIP_SYMBOL("c_clusters"),clusters[0],nclusters*nfeatures*sizeof(float),0,hipMemcpyHostToDevice);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  transferTime += elapsedTime * 1.e-3; // convert to seconds


    /* setup execution parameters.
	   changed to 2d (source code on NVIDIA CUDA Programming Guide) */
    dim3  grid( num_blocks_perdim, num_blocks_perdim );
    dim3  threads( num_threads_perdim*num_threads_perdim );
    
	/* execute the kernel */
    hipEventRecord(start, 0);
    kmeansPoint<<< grid, threads >>>( feature_d,
                                      nfeatures,
                                      npoints,
                                      nclusters,
                                      membership_d,
                                      clusters_d,
									  block_clusters_d,
									  block_deltas_d);

	hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    kernelTime += elapsedTime * 1.e-3;
    //CHECK_CUDA_ERROR();

  hipEventRecord(start, 0);
	/* copy back membership (device to host) */
#ifdef UNIFIED_MEMORY
	//CUDA_SAFE_CALL(hipMemcpy(membership_new, membership_d, npoints*sizeof(int), hipMemcpyDeviceToHost));	
#else
	hipMemcpy(membership_new, membership_d, npoints*sizeof(int), hipMemcpyDeviceToHost);	
#endif
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  transferTime += elapsedTime * 1.e-3; // convert to seconds

#ifdef BLOCK_CENTER_REDUCE
    /*** Copy back arrays of per block sums ***/
#ifdef UNIFIED_MEMORY
    float * block_clusters_h = NULL;
#else
    float * block_clusters_h = (float *) malloc(
        num_blocks_perdim * num_blocks_perdim * 
        nclusters * nfeatures * sizeof(float));
#endif
        
  hipEventRecord(start, 0);
#ifdef UNIFIED_MEMORY
  /*
    CUDA_SAFE_CALL(hipMemcpy(block_clusters_h, block_clusters_d, 
        num_blocks_perdim * num_blocks_perdim * 
        nclusters * nfeatures * sizeof(float), 
        hipMemcpyDeviceToHost));
        */
  block_clusters_h = block_clusters_d;
#else
	hipMemcpy(block_clusters_h, block_clusters_d, 
        num_blocks_perdim * num_blocks_perdim * 
        nclusters * nfeatures * sizeof(float), 
        hipMemcpyDeviceToHost);
#endif
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  transferTime += elapsedTime * 1.e-3; // convert to seconds
#endif
#ifdef BLOCK_DELTA_REDUCE
#ifdef UNIFIED_MEMORY
  int * block_deltas_h = NULL;
#else
    int * block_deltas_h = (int *) malloc(
        num_blocks_perdim * num_blocks_perdim * sizeof(int));
#endif
        
  hipEventRecord(start, 0);
#ifdef UNIFIED_MEMORY
  block_deltas_h = block_deltas_d;
  /*
    CUDA_SAFE_CALL(hipMemcpy(block_deltas_h, block_deltas_d, 
        num_blocks_perdim * num_blocks_perdim * sizeof(int), 
        hipMemcpyDeviceToHost));
        */
#else
	hipMemcpy(block_deltas_h, block_deltas_d, 
        num_blocks_perdim * num_blocks_perdim * sizeof(int), 
        hipMemcpyDeviceToHost);
#endif
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  transferTime += elapsedTime * 1.e-3; // convert to seconds
#endif
    
	/* for each point, sum data points in each cluster
	   and see if membership has changed:
	     if so, increase delta and change old membership, and update new_centers;
	     otherwise, update new_centers */
	delta = 0;
	for (i = 0; i < npoints; i++)
	{		
		int cluster_id = membership_new[i];
		new_centers_len[cluster_id]++;
		if (membership_new[i] != membership[i])
		{
#ifdef CPU_DELTA_REDUCE
			delta++;
#endif
			membership[i] = membership_new[i];
		}
#ifdef CPU_CENTER_REDUCE
		for (j = 0; j < nfeatures; j++)
		{			
			new_centers[cluster_id][j] += feature[i][j];
		}
#endif
	}
	

#ifdef BLOCK_DELTA_REDUCE	
    /*** calculate global sums from per block sums for delta and the new centers ***/    
	
	//debug
    for(i = 0; i < num_blocks_perdim * num_blocks_perdim; i++) {
        delta += block_deltas_h[i];
    }
        
#endif
#ifdef BLOCK_CENTER_REDUCE	
	
	for(int j = 0; j < nclusters;j++) {
		for(int k = 0; k < nfeatures;k++) {
			block_new_centers[j*nfeatures + k] = 0.f;
		}
	}

    for(i = 0; i < num_blocks_perdim * num_blocks_perdim; i++) {
		for(int j = 0; j < nclusters;j++) {
			for(int k = 0; k < nfeatures;k++) {
				block_new_centers[j*nfeatures + k] += block_clusters_h[i * nclusters*nfeatures + j * nfeatures + k];
			}
		}
    }
	
#ifdef BLOCK_CENTER_REDUCE
	for(int j = 0; j < nclusters;j++) {
		for(int k = 0; k < nfeatures;k++)
			new_centers[j][k]= block_new_centers[j*nfeatures + k];		
	}
#endif

#endif

	return delta;
	
}
/* ------------------- kmeansCuda() end ------------------------ */    
