#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////////////////////////
// file:	altis\src\cuda\level2\kmeans\kmeans_cuda.cu
//
// summary:	Kmeans cuda class
// 
// origin:  Rodinia(http://rodinia.cs.virginia.edu/doku.php)
//
////////////////////////////////////////////////////////////////////////////////////////////////////

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#define THREADS_PER_DIM 16
#define BLOCKS_PER_DIM 16
#define THREADS_PER_BLOCK THREADS_PER_DIM*THREADS_PER_DIM

#include "cudacommon.h"
#include "ResultDatabase.h"
#include "OptionParser.h"
#include "kmeans_cuda_kernel.cu"

#include "kmeans.h"

//#define BLOCK_DELTA_REDUCE
//#define BLOCK_CENTER_REDUCE

#define CPU_DELTA_REDUCE
#define CPU_CENTER_REDUCE

int setup(ResultDatabase &resultDB, OptionParser &op);

// GLOBAL!!!!!
/* sqrt(256) -- see references for this choice */
unsigned int num_threads_perdim = THREADS_PER_DIM;					
/* temporary */
unsigned int num_blocks_perdim = BLOCKS_PER_DIM;
/* number of threads */
unsigned int num_threads = num_threads_perdim*num_threads_perdim;	
/* number of blocks */
unsigned int num_blocks = num_blocks_perdim*num_blocks_perdim;		

/* _d denotes it resides on the device */
/* newly assignment membership */
int    *membership_new;												
/* inverted data array */
float  *feature_d;													
/* original (not inverted) data array */
float  *feature_flipped_d;											
/* membership on the device */
int    *membership_d;												
/* sum of points in a cluster (per block) */
float  *block_new_centers;											
/* cluster centers on the device */
float  *clusters_d;													
/* per block calculation of cluster centers */
float  *block_clusters_d;											
/* per block calculation of deltas */
int    *block_deltas_d;												


////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Adds a benchmark specifier options. </summary>
///
/// <remarks>	Edward Hu (bodunhu@utexas.edu) 5/20/2020. </remarks>
///
/// <param name="op">	[in,out] The operation. </param>
////////////////////////////////////////////////////////////////////////////////////////////////////

void addBenchmarkSpecOptions(OptionParser &op) {
    op.addOption("maxClusters", OPT_INT, "5", "maximum number of clusters allowed");
    op.addOption("minClusters", OPT_INT, "5", "minimum number of clusters allowed");
    op.addOption("threshold", OPT_FLOAT, "0.001", "threshold value");
    op.addOption("loops", OPT_INT, "1", "loop for each number of clusters");
    op.addOption("rmse", OPT_BOOL, "0", "calculate RMSE (default off)");
    op.addOption("outputCenters", OPT_BOOL, "0", "output cluster center coordinates (default off)");
    op.addOption("iterations", OPT_INT, "1000", "number of kmeans operation for each number of cluster in each loop");
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Executes the benchmark operation. </summary>
///
/// <remarks>	Edward Hu (bodunhu@utexas.edu) 5/20/2020. </remarks>
///
/// <param name="resultDB">	[in,out] The result database. </param>
/// <param name="op">	   	[in,out] The operation. </param>
////////////////////////////////////////////////////////////////////////////////////////////////////

void RunBenchmark(ResultDatabase &resultDB, OptionParser &op) {
    printf("Running KMeans\n");
    setup(resultDB, op);
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	allocate device memory, calculate number of blocks and threads, and invert the data array. </summary>
///
/// <remarks>	Edward Hu (bodunhu@utexas.edu) 5/20/2020. </remarks>
///
/// <param name="npoints">  	The npoints. </param>
/// <param name="nfeatures">	The nfeatures. </param>
/// <param name="nclusters">	The nclusters. </param>
/// <param name="features"> 	[in,out] If non-null, the features. </param>
////////////////////////////////////////////////////////////////////////////////////////////////////

void allocateMemory(int npoints, int nfeatures, int nclusters, float **features)
{	
	num_blocks = npoints / num_threads;
	if (npoints % num_threads > 0)		/* defeat truncation */
		num_blocks++;

	num_blocks_perdim = sqrt((double) num_blocks);
	while (num_blocks_perdim * num_blocks_perdim < num_blocks)	// defeat truncation (should run once)
		num_blocks_perdim++;

	num_blocks = num_blocks_perdim*num_blocks_perdim;

	/* allocate memory for memory_new[] and initialize to -1 (host) */

    membership_new = (int*) malloc(npoints * sizeof(int));
    assert(membership_new);
	for(int i=0;i<npoints;i++) {
		membership_new[i] = -1;
	}

	/* allocate memory for block_new_centers[] (host) */
	block_new_centers = (float *) malloc(nclusters*nfeatures*sizeof(float));
    assert(block_new_centers);
    
	/* allocate memory for feature_flipped_d[][], feature_d[][] (device) */
	checkCudaErrors(hipMalloc((void**) &feature_flipped_d, npoints*nfeatures*sizeof(float)));
	checkCudaErrors(hipMemcpy(feature_flipped_d, features[0], npoints*nfeatures*sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**) &feature_d, npoints*nfeatures*sizeof(float)));
		
	/* invert the data array (kernel execution) */	
	invert_mapping<<<num_blocks,num_threads>>>(feature_flipped_d,feature_d,npoints,nfeatures);
		
	/* allocate memory for membership_d[] and clusters_d[][] (device) */
	checkCudaErrors(hipMalloc((void**) &membership_d, npoints*sizeof(int)));
	checkCudaErrors(hipMalloc((void**) &clusters_d, nclusters*nfeatures*sizeof(float)));
	
#ifdef BLOCK_DELTA_REDUCE
	// allocate array to hold the per block deltas on the gpu side
	checkCudaErrors(hipMalloc((void**) &block_deltas_d, num_blocks_perdim * num_blocks_perdim * sizeof(int)));
	checkCudaErrors(hipMemcpy(block_delta_d, &delta_h, sizeof(int), hipMemcpyHostToDevice));
#endif

#ifdef BLOCK_CENTER_REDUCE
	// allocate memory and copy to card cluster  array in which to accumulate center points for the next iteration
    checkCudaErrors(hipMalloc((void**) &block_clusters_d, 
            num_blocks_perdim * num_blocks_perdim * 
            nclusters * nfeatures * sizeof(float)));
    checkCudaErrors(hipMemcpy(new_clusters_d, new_centers[0], nclusters*nfeatures*sizeof(float), hipMemcpyHostToDevice));
#endif

}
/* -------------- allocateMemory() end ------------------- */


////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	free host and device memory. </summary>
///
/// <remarks>	Edward Hu (bodunhu@utexas.edu) 5/20/2020. </remarks>
////////////////////////////////////////////////////////////////////////////////////////////////////

void deallocateMemory()
{
	free(membership_new);
	free(block_new_centers);
    checkCudaErrors(hipFree(membership_d));
	checkCudaErrors(hipFree(clusters_d));
	checkCudaErrors(hipFree(feature_d));
	//hipFree(feature_flipped_d);
	//hipFree(membership_d);

#ifdef BLOCK_CENTER_REDUCE
    checkCudaErrors(hipFree(block_clusters_d));
#endif
#ifdef BLOCK_DELTA_REDUCE
    checkCudaErrors(hipFree(block_deltas_d));
#endif
}
/* -------------- deallocateMemory() end ------------------- */


/* ------------------- kmeansCuda() ------------------------ */    

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Kmeans cuda. </summary>
///
/// <remarks>	Edward Hu (bodunhu@utexas.edu) 5/20/2020. </remarks>
///
/// <param name="feature">		  	[in,out] If non-null, the feature. </param>
/// <param name="nfeatures">	  	The nfeatures. </param>
/// <param name="npoints">		  	The npoints. </param>
/// <param name="nclusters">	  	The nclusters. </param>
/// <param name="membership">	  	[in,out] If non-null, the membership. </param>
/// <param name="clusters">		  	[in,out] If non-null, the clusters. </param>
/// <param name="new_centers_len">	[in,out] If non-null, length of the new centers. </param>
/// <param name="new_centers">	  	[in,out] If non-null, the new centers. </param>
/// <param name="transferTime">   	[in,out] The transfer time. </param>
/// <param name="kernelTime">	  	[in,out] The kernel time. </param>
/// <param name="resultDB">		  	[in,out] The result database. </param>
///
/// <returns>	An int. </returns>
////////////////////////////////////////////////////////////////////////////////////////////////////

int	// delta -- had problems when return value was of float type
kmeansCuda(float  **feature,				/* in: [npoints][nfeatures] */
           int      nfeatures,				/* number of attributes for each point */
           int      npoints,				/* number of data points */
           int      nclusters,				/* number of clusters */
           int     *membership,				/* which cluster the point belongs to */
		   float  **clusters,				/* coordinates of cluster centers */
		   int     *new_centers_len,		/* number of elements in each cluster */
           float  **new_centers,			/* sum of elements in each cluster */
           double &transferTime,
           double &kernelTime,
		   ResultDatabase &resultDB)
{
	int delta = 0;			/* if point has moved */
	int i,j;				/* counters */

    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    float elapsedTime;

    checkCudaErrors(hipEventRecord(start, 0));
    /* copy membership (host to device) */
    checkCudaErrors(hipMemcpy(membership_d, membership_new, npoints*sizeof(int), hipMemcpyHostToDevice));
    /* copy clusters (host to device) */
    checkCudaErrors(hipMemcpy(clusters_d, clusters[0], nclusters*nfeatures*sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
    transferTime += elapsedTime * 1.e-3; // convert to seconds

    hipError_t err;

	/* set up texture */
    hipChannelFormatDesc chDesc0 = hipCreateChannelDesc<float>();
    t_features.filterMode = hipFilterModePoint;   
    t_features.normalized = false;
    t_features.channelDesc = chDesc0;

	err = hipBindTexture(NULL, &t_features, feature_d, &chDesc0, npoints*nfeatures*sizeof(float));
    if (err != hipSuccess) {
        printf("Error: Couldn't bind features array to texture, %d", err);
        exit(0);
    }

	hipChannelFormatDesc chDesc1 = hipCreateChannelDesc<float>();
    t_features_flipped.filterMode = hipFilterModePoint;   
    t_features_flipped.normalized = false;
    t_features_flipped.channelDesc = chDesc1;

	err = hipBindTexture(NULL, &t_features_flipped, feature_flipped_d, &chDesc1, npoints*nfeatures*sizeof(float));
    if (err != hipSuccess) {
        printf("Error: Couldn't bind features_flipped array to texture, %d", err);
        exit(0);
    }
	// hipChannelFormatDesc chDesc2 = hipCreateChannelDesc<float>();
    // t_clusters.filterMode = hipFilterModePoint;   
    // t_clusters.normalized = false;
    // t_clusters.channelDesc = chDesc2;

	// err = hipBindTexture(NULL, &t_clusters, clusters_d, &chDesc2, nclusters*nfeatures*sizeof(float));
    // if(err != hipSuccess) {
    //     printf("Error: Couldn't bind clusters array to texture, %d", err);
    //     exit(0);
    // }

    checkCudaErrors(hipEventRecord(start, 0));
	/* copy clusters to constant memory */
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_clusters), clusters[0], 
                nclusters * nfeatures * sizeof(float),
                0, hipMemcpyHostToDevice));
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
    transferTime += elapsedTime * 1.e-3; // convert to seconds


    /* setup execution parameters.
	   changed to 2d (source code on NVIDIA CUDA Programming Guide) */
    dim3  grid( num_blocks_perdim, num_blocks_perdim );
    dim3  threads( num_threads_perdim*num_threads_perdim );
    
	/* execute the kernel */
    checkCudaErrors(hipEventRecord(start, 0));
    kmeansPoint<<< grid, threads >>>( feature_d,
                                      nfeatures,
                                      npoints,
                                      nclusters,
                                      membership_d,
                                      clusters_d,
									  block_clusters_d,
									  block_deltas_d);

    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
    kernelTime += elapsedTime * 1.e-3;

    checkCudaErrors(hipEventRecord(start, 0));
	/* copy back membership (device to host) */
	checkCudaErrors(hipMemcpy(membership_new, membership_d, npoints*sizeof(int), hipMemcpyDeviceToHost));	
  
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
    transferTime += elapsedTime * 1.e-3; // convert to seconds

#ifdef BLOCK_CENTER_REDUCE
    /*** Copy back arrays of per block sums ***/
    float * block_clusters_h = (float *) malloc(
        num_blocks_perdim * num_blocks_perdim * 
        nclusters * nfeatures * sizeof(float));
        
    checkCudaErrors(hipEventRecord(start, 0));

	checkCudaErrors(hipMemcpy(block_clusters_h, block_clusters_d, 
        num_blocks_perdim * num_blocks_perdim * 
        nclusters * nfeatures * sizeof(float), 
        hipMemcpyDeviceToHost));

    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
    transferTime += elapsedTime * 1.e-3; // convert to seconds
#endif
#ifdef BLOCK_DELTA_REDUCE
    int * block_deltas_h = (int *) malloc(
        num_blocks_perdim * num_blocks_perdim * sizeof(int));
    assert(block_deltas_h);   
    checkCudaErrors(hipEventRecord(start, 0));

	checkCudaErrors(hipMemcpy(block_deltas_h, block_deltas_d, 
        num_blocks_perdim * num_blocks_perdim * sizeof(int), 
        hipMemcpyDeviceToHost));

    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
    transferTime += elapsedTime * 1.e-3; // convert to seconds
#endif
    
	/* for each point, sum data points in each cluster
	   and see if membership has changed:
	     if so, increase delta and change old membership, and update new_centers;
	     otherwise, update new_centers */
	delta = 0;
	for (i = 0; i < npoints; i++)
	{		
		int cluster_id = membership_new[i];
		new_centers_len[cluster_id]++;
		if (membership_new[i] != membership[i])
		{
#ifdef CPU_DELTA_REDUCE
			delta++;
#endif
			membership[i] = membership_new[i];
		}
#ifdef CPU_CENTER_REDUCE
		for (j = 0; j < nfeatures; j++)
		{			
			new_centers[cluster_id][j] += feature[i][j];
		}
#endif
	}
	

#ifdef BLOCK_DELTA_REDUCE	
    /*** calculate global sums from per block sums for delta and the new centers ***/    
	
	//debug
    for(i = 0; i < num_blocks_perdim * num_blocks_perdim; i++) {
        delta += block_deltas_h[i];
    }
        
#endif
#ifdef BLOCK_CENTER_REDUCE	
	
	for(int j = 0; j < nclusters;j++) {
		for(int k = 0; k < nfeatures;k++) {
			block_new_centers[j*nfeatures + k] = 0.f;
		}
	}

    for(i = 0; i < num_blocks_perdim * num_blocks_perdim; i++) {
		for(int j = 0; j < nclusters;j++) {
			for(int k = 0; k < nfeatures;k++) {
				block_new_centers[j*nfeatures + k] += block_clusters_h[i * nclusters*nfeatures + j * nfeatures + k];
			}
		}
    }
	
#ifdef BLOCK_CENTER_REDUCE
	for(int j = 0; j < nclusters;j++) {
		for(int k = 0; k < nfeatures;k++)
			new_centers[j][k]= block_new_centers[j*nfeatures + k];		
	}
#endif

#endif

	return delta;
	
}
/* ------------------- kmeansCuda() end ------------------------ */    
