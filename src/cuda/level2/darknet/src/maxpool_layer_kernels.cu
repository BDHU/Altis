#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "maxpool_layer.h"
#include "hip/hip_runtime.h"
}

extern "C" void forward_maxpool_layer_gpu(maxpool_layer layer, network net)
{
    float one = 1;
    hipdnnStatus_t stat = hipdnnPoolingForward(cudnn_handle(), layer.poolingDesc, &one,
            layer.poolingInputTensorDesc, net.input_gpu, &one, layer.poolingOutputTensorDesc,
            layer.output_gpu);
    assert(stat == HIPDNN_STATUS_SUCCESS);
}

extern "C" void backward_maxpool_layer_gpu(maxpool_layer layer, network net)
{
    float one = 1;
    float zero = 0;
    hipdnnStatus_t stat = hipdnnPoolingBackward(cudnn_handle(), layer.poolingDesc, &one,
            layer.poolingOutputTensorDesc,
            layer.output_gpu, layer.poolingOutputTensorDesc, layer.dy,
            layer.poolingInputTensorDesc, net.input_gpu, &zero, layer.poolingInputTensorDesc,
            layer.delta_gpu);
    assert(stat == HIPDNN_STATUS_SUCCESS);
}

