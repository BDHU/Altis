#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <string>
#include <math.h>
#include <cassert>
#include <hip/hip_runtime.h>

#include "OptionParser.h"
#include "ResultDatabase.h"
#include "cudacommon.h"
#include "fdtd2d.h"

#define DEFAULT_GPU 0

// TODO subject to change
#define DIM_THREAD_BLOCK_X 32
#define DIM_THREAD_BLOCK_Y 8

#define SMALL_FLOAT_VAL 0.00000001f
#define PERCENT_DIFF_ERROR_THRESHOLD 10.05

#define tmax 500
#define NX 2048
#define NY 2048


float absVal(float a);
float percentDiff(double val1, double val2);

using namespace std;


float absVal(float a) {
    if (a < 0) return (-1*a);
    else       return a;
}

float percentDiff(double val1, double val2) {
    if ((absVal(val1) < 0.01) && (absVal(val2) < 0.01))
        return 0.0f;
    else
        return 100.0f * (absVal(absVal(val1 - val2) / absVal(val1 + SMALL_FLOAT_VAL)));
} 


void addBenchmarkSpecOptions(OptionParser &op) {
   // TODO, maybe add benchmark specs 
}

void init_arrays(DATA_TYPE *_fict_, DATA_TYPE *ex,
        DATA_TYPE *ey, DATA_TYPE *hz) {
    assert(_fict_ && ex && ey && hz);

    int i = 0;
    for (; i < tmax; i++) {
        _fict_[i] = (DATA_TYPE)i;
    }

    int j;
    for (i = 0; i < NX; i++) {
        for (j = 0; j < NY; j++) {
            int index_to_update = i * NY + j;
            ex[index_to_update] = ((DATA_TYPE)i*(j+1)+1) / NX;
            ey[index_to_update] = ((DATA_TYPE)(i-1)*(j+2)+2) / NX;
            hz[index_to_update] = ((DATA_TYPE)(i-9)*(j+4)+3) / NX;
        }
    }
}

void run_fdtd_cpu(DATA_TYPE* _fict_, DATA_TYPE* ex, DATA_TYPE* ey, DATA_TYPE* hz)
{
    assert(_fict_ && ex && ey && hz);
	int t, i, j;

	for (t=0; t < tmax; t++)
	{
		for (j=0; j < NY; j++)
		{
			ey[0*NY + j] = _fict_[t];
		}

		for (i = 1; i < NX; i++)
		{
       		for (j = 0; j < NY; j++)
			{
       			ey[i*NY + j] = ey[i*NY + j] - 0.5*(hz[i*NY + j] - hz[(i-1)*NY + j]);
        	}
		}

		for (i = 0; i < NX; i++)
		{
       		for (j = 1; j < NY; j++)
			{
				ex[i*(NY+1) + j] = ex[i*(NY+1) + j] - 0.5*(hz[i*NY + j] - hz[i*NY + (j-1)]);
			}
		}

		for (i = 0; i < NX; i++)
		{
			for (j = 0; j < NY; j++)
			{
				hz[i*NY + j] = hz[i*NY + j] - 0.7*(ex[i*(NY+1) + (j+1)] - ex[i*(NY+1) + j] + ey[(i+1)*NY + j] - ey[i*NY + j]);
			}
		}
	}
}


void compareResults(DATA_TYPE* hz1, DATA_TYPE* hz2)
{
	int i, j, fail;
	fail = 0;

	for (i=0; i < NX; i++)
	{
		for (j=0; j < NY; j++)
		{
			if (percentDiff(hz1[i*NY + j], hz2[i*NY + j]) > PERCENT_DIFF_ERROR_THRESHOLD)
			{
				fail++;
			}
		}
	}

	// Print results
	cout << "Non-Matching CPU-GPU Outputs Beyond Error Threshold of " <<
        PERCENT_DIFF_ERROR_THRESHOLD << " Percent: " << fail << endl;
}

__global__ void kernel1(DATA_TYPE* _fict_, DATA_TYPE *ex, DATA_TYPE *ey, DATA_TYPE *hz, int t)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < NX) && (j < NY))
	{
		if (i == 0)
		{
			ey[i * NY + j] = _fict_[t];
		}
		else
		{
			ey[i * NY + j] = ey[i * NY + j] - 0.5f*(hz[i * NY + j] - hz[(i-1) * NY + j]);
		}
	}
}

__global__ void kernel2(DATA_TYPE *ex, DATA_TYPE *ey, DATA_TYPE *hz, int t)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < NX) && (j < NY) && (j > 0))
	{
		ex[i * (NY+1) + j] = ex[i * (NY+1) + j] - 0.5f*(hz[i * NY + j] - hz[i * NY + (j-1)]);
	}
}

__global__ void kernel3(DATA_TYPE *ex, DATA_TYPE *ey, DATA_TYPE *hz, int t)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < NX) && (j < NY))
	{
		hz[i * NY + j] = hz[i * NY + j] - 0.7f*(ex[i * (NY+1) + (j+1)] - ex[i * (NY+1) + j] + ey[(i + 1) * NY + j] - ey[i * NY + j]);
	}
}

void run_fdtd_cuda(DATA_TYPE *_fict_, DATA_TYPE *ex, DATA_TYPE *ey, DATA_TYPE *hz, DATA_TYPE *hz_from_gpu) {
    assert(_fict_ && ex && ey && hz && hz_from_gpu);

    DATA_TYPE *_fict_gpu = NULL;
    DATA_TYPE *ex_gpu = NULL;
    DATA_TYPE *ey_gpu = NULL;
    DATA_TYPE *hz_gpu = NULL;

    // allocating resources
    hipMalloc((void **)&_fict_gpu, sizeof(DATA_TYPE) * tmax);
    if (hipGetLastError() != hipSuccess)
    {
        cout << "hipMalloc d_A returned error code" << endl;
        exit(1);
    }

	hipMalloc((void **)&ex_gpu, sizeof(DATA_TYPE) * NX * (NY + 1));
    if (hipGetLastError() != hipSuccess)
    {
        cout << "hipMalloc d_A returned error code" << endl;
        exit(1);
    }


	hipMalloc((void **)&ey_gpu, sizeof(DATA_TYPE) * (NX + 1) * NY);
    if (hipGetLastError() != hipSuccess)
    {
        cout << "hipMalloc d_A returned error code" << endl;
        exit(1);
    }

	hipMalloc((void **)&hz_gpu, sizeof(DATA_TYPE) * NX * NY);
    if (hipGetLastError() != hipSuccess)
    {
        cout << "hipMalloc d_A returned error code" << endl;
        exit(1);
    }

    // copy data to device
	hipMemcpy(_fict_gpu, _fict_, sizeof(DATA_TYPE) * tmax, hipMemcpyHostToDevice);
    if (hipGetLastError() != hipSuccess)
    {
        cout << "hipMemcpy d_A returned error code" << endl;
        exit(1);
    }

	hipMemcpy(ex_gpu, ex, sizeof(DATA_TYPE) * NX * (NY + 1), hipMemcpyHostToDevice);
    if (hipGetLastError() != hipSuccess)
    {
        cout << "hipMalloc d_A returned error code" << endl;
        exit(1);
    }

	hipMemcpy(ey_gpu, ey, sizeof(DATA_TYPE) * (NX + 1) * NY, hipMemcpyHostToDevice);
    if (hipGetLastError() != hipSuccess)
    {
        cout << "hipMalloc d_A returned error code" << endl;
        exit(1);
    }

	hipMemcpy(hz_gpu, hz, sizeof(DATA_TYPE) * NX * NY, hipMemcpyHostToDevice);
    if (hipGetLastError() != hipSuccess)
    {
        cout << "hipMalloc d_A returned error code" << endl;
        exit(1);
    }

    //TODO: subject to change
    dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
    dim3 grid( (size_t)ceil(((float)NY) / ((float)block.x)),
            (size_t)ceil(((float)NX) / ((float)block.y)));

    // without hyperq or graph
    // TODO could use two streams to overlap execution
    int t = 0;
    for (; t < tmax; t++) {
        kernel1<<<grid, block>>>(_fict_gpu, ex_gpu, ey_gpu, hz_gpu, t);
        //hipDeviceSynchronize();
        kernel2<<<grid, block>>>(ex_gpu, ey_gpu, hz_gpu, t);
        hipDeviceSynchronize();
        kernel3<<<grid, block>>>(ex_gpu, ey_gpu, hz_gpu, t);
        hipDeviceSynchronize();
    }
    hipMemcpy(hz_from_gpu, hz_gpu, sizeof(DATA_TYPE) * NX * NY, hipMemcpyDeviceToHost);
    if (hipGetLastError() != hipSuccess) {
        cout << "can't copy results back to host, error code " << hipGetLastError() << endl;
        exit(1);
    }
    hipFree(_fict_gpu);
    hipFree(ex_gpu);
    hipFree(ey_gpu);
    hipFree(hz_gpu);
    // TODO, we are using cuda graph now, it might be supported at the sametime
    // with unified memory, but worry about later
    
    
}

void RunBenchmark(ResultDatabase &DB, OptionParser &op) {
    cout << "Running FDTD" << endl;
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    if (deviceProp.computeMode == hipComputeModeProhibited) {
        cout << "Error: deivce in <Compute Mode Prohibited>,"
            << "no threads can use ::hipSetDevice()." << endl;
        exit(1);
    }
    if (hipGetLastError() != hipSuccess) {
        cout << "hipGetDeviceProperties returns error code" << endl;
    }
    hipSetDevice(DEFAULT_GPU);


    bool quiet = op.getOptionBool("quiet");
    int passes = op.getOptionInt("passes");

    // allocating resources
    DATA_TYPE *_fict_cpu = NULL;
    DATA_TYPE *ex_cpu = NULL;
    DATA_TYPE *ey_cpu = NULL;
    DATA_TYPE *hz_cpu = NULL;
    DATA_TYPE *hz_from_gpu = NULL;

    // alloc without unified mem
    _fict_cpu = (DATA_TYPE *)malloc(tmax * sizeof(DATA_TYPE));
    ex_cpu = (DATA_TYPE *)malloc(NX * (NY + 1) * sizeof(DATA_TYPE));
    ey_cpu = (DATA_TYPE *)malloc((NX + 1) * NY * sizeof(DATA_TYPE));
    hz_cpu = (DATA_TYPE *)malloc(NX * NY * sizeof(DATA_TYPE));
    cout << "alloced about " << NX << " bytes" << endl;
    hz_from_gpu = (DATA_TYPE *)malloc(NX * NY * sizeof(DATA_TYPE));
    
    srand(1);
    init_arrays(_fict_cpu, ex_cpu, ey_cpu, hz_cpu);

    int pass = 0;
    for (; pass < 1; pass++) {
        run_fdtd_cuda(_fict_cpu, ex_cpu, ey_cpu, hz_cpu, hz_from_gpu);
    }

    // TODO may not necessary
    srand(1);
    init_arrays(_fict_cpu, ex_cpu, ey_cpu, hz_cpu);

    for (pass = 0; pass < 1; pass ++) {
        run_fdtd_cpu(_fict_cpu, ex_cpu, ey_cpu, hz_cpu);
    }
    compareResults(hz_cpu, hz_from_gpu);
    // clean up
    free(_fict_cpu);
    free(ex_cpu);
    free(ey_cpu);
    free(hz_cpu);
    free(hz_from_gpu);
}
