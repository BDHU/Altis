#include "fdtd.h"

using namespace std;

void RunBenchmark(ResultDATAbase &result DB, OptionParser &op) {
    cout << "Running FDTD" << endl;
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
}
