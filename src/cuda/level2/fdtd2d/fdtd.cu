#include "hip/hip_runtime.h"
#include "fdtd.h"

// TODO subject to change
#define DIM_THREAD_BLOCK_X 32
#define DIM_THREAD_BLOCK_Y 8

using namespace std;

void addBenchmarkSpecOptions(OptionParser *op) {
   // TODO, maybe add benchmark specs 
}

void init_arrays(DATA_TYPE *_fict_, DATA_TYPE *ex,
        DATA_TYPE *ey, DATA_TYPE *hz) {
    assert(_fict_ && ex && ey && hz);

    int i = 0
    for (; i < tmax; i++) {
        _fict_[i] = (DATA_TYPE)i;
    }

    int j;
    for (i = 0; i < NX; i++) {
        for (j = 0; j < NY; j++) {
            int index_to_update = i * NY + j;
            ex[index_to_update] = ((DATA_TYPE)i*(j+1)+1) / NX;
            ey[index_to_update] = ((DATA_TYPE)(i-1)*(j+2)+2) / NX;
            hz[index_to_update] = ((DATA_TYPE)(i-9)*(j+4)+3) / NX;
        }
    }
}

void run_fdtd_cuda(DATA_TYPE *_fict_, DATA_TYPE *ex, DATA_TYPE *ey, DATA_TYPE *hz, DATA_TYPE *hz_from_gpu) {
    assert(_fict_ && ex && ey && hz && hz_from_gpu);

#ifndef UNIFIED_MEMORY
    DATA_TYPE *_fict_gpu = NULL;
    DATA_TYPE *ex_gpu = NULL;
    DATA_TYPE *ey_gpu = NULL;
    DATA_TYPE *hz_gpu = NULL;

    // allocating resources
    hipMalloc((void **)&_fict_gpu, sizeof(DATA_TYPE) * tmax);
    if (hipGetLastError() != hipSuccess)
    {
        cout << "hipMalloc d_A returned error code" << endl;
        exit(1);
    }

	hipMalloc((void **)&ex_gpu, sizeof(DATA_TYPE) * NX * (NY + 1));
    if (hipGetLastError() != hipSuccess)
    {
        cout << "hipMalloc d_A returned error code" << endl;
        exit(1);
    }

	hipMalloc((void **)&ey_gpu, sizeof(DATA_TYPE) * (NX + 1) * NY);
    if (hipGetLastError() != hipSuccess)
    {
        cout << "hipMalloc d_A returned error code" << endl;
        exit(1);
    }

	hipMalloc((void **)&hz_gpu, sizeof(DATA_TYPE) * NX * NY);
    if (hipGetLastError() != hipSuccess)
    {
        cout << "hipMalloc d_A returned error code" << endl;
        exit(1);
    }

    // copy data to device
	hipMemcpy(_fict_gpu, _fict_, sizeof(DATA_TYPE) * tmax, hipMemcpyHostToDevice);
    if (hipGetLastError() != hipSuccess)
    {
        cout << "hipMemcpy d_A returned error code" << endl;
        exit(1);
    }

	hipMemcpy(ex_gpu, ex, sizeof(DATA_TYPE) * NX * (NY + 1), hipMemcpyHostToDevice);
    if (hipGetLastError() != hipSuccess)
    {
        cout << "hipMalloc d_A returned error code" << endl;
        exit(1);
    }

	hipMemcpy(ey_gpu, ey, sizeof(DATA_TYPE) * (NX + 1) * NY, hipMemcpyHostToDevice);
    if (hipGetLastError() != hipSuccess)
    {
        cout << "hipMalloc d_A returned error code" << endl;
        exit(1);
    }

	hipMemcpy(hz_gpu, hz, sizeof(DATA_TYPE) * NX * NY, hipMemcpyHostToDevice);
    if (hipGetLastError() != hipSuccess)
    {
        cout << "hipMalloc d_A returned error code" << endl;
        exit(1);
    }

    //TODO: subject to change
    dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
    dim3 grid( (size_t)ceil(((float)NY) / ((float)block.x)),
            (size_t)ceil(((float)NX) / ((float)block.y)));

    int t = 0;
    for (; t < tmax; t++) {
        kernel1<<<grid, block>>>(_fict_gpu, ex_gpu, ey_gpu, hz_gpu, t);
    }

#else

#endif

}

void RunBenchmark(ResultDatabase &result DB, OptionParser &op) {
    cout << "Running FDTD" << endl;
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    if (deviceProp.computeMode == hipComputeModeProhibited) {
        cout << "Error: deivce in <Compute Mode Prohibited>,"
            << "no threads can use ::hipSetDevice()." << endl;
        exit(1);
    }
    if (hipGetLastError() != hipSuccess) {
        cout << "hipGetDeviceProperties returns error code" << endl;
    }
    hipSetDevice(DEFAULT_GPU);


    bool quiet = op.getOptionBool("quiet");
    int passes = op.getOptionInt("passes");

#ifndef UNIFIED_MEMORY
    // allocating resources
    DATA_TYPE *_fict_cpu = NULL;
    DATA_TYPE *ex_cpu = NULL;
    DATA_TYPE *ey_cpu = NULL;
    DATA_TYPE *hz_cpu = NULL;
    DATA_TYPE *hz_from_gpu = NULL;

    // alloc without unified mem
    _fict_cpu = (DATA_TYPE *)malloc(tmax * sizeof(DATA_TYPE));
    ex_cpu = (DATA_TYPE *)malloc(NX * (NY + 1) * sizeof(DATA_TYPE));
    ey_cpu = (DATA_TYPE *)malloc((NX + 1) * NY * sizeof(DATA_TYPE));
    hz_cpu = (DATA_TYPE *)malloc(NX * NY * sizeof(DATA_TYPE));
    hz_from_gpu = (DATA_TYPE *)malloc(NX * NY * sizeof(DATA_TYPE));
    srand(1);
    init_arrays(_fict_cpu, ex_cpu, ey_cpu, hz_cpu);

    int pass = 0;
    for (; pass < passes; pass++) {
        run_fdtd_cuda(_fict_cpu, ex_cpu, ey_cpu, hz_cpu, hz_gpu);
    }
#else
    DATA_TYPE *_fict_ = NULL;
    DATA_TYPE *ex = NULL;
    DATA_TYPE *ey = NULL;
    DATA_TYPE *hz_cpu = NULL;
    DATA_TYPE *hz_gpu = NULL;
    hipMallocManaged(&_fict_, tmax * sizeof(DATA_TYPE));
    if (hipGetLastError() != hipSuccess) {
        cout << "cudaMallocManged failed" << endl;
    }
    hipMallocManaged(&ex, NX * (NY + 1) * sizeof(DATA_TYPE));
    if (hipGetLastError() != hipSuccess) {
        cout << "cudaMallocManged failed" << endl;
    }
    hipMallocManaged(&ey, (NX + 1) * NY * sizeof(DATA_TYPE));
    if (hipGetLastError() != hipSuccess) {
        cout << "cudaMallocManged failed" << endl;
    }
    hipMallocManaged(&hz_cpu, NX * NY * sizeof(DATA_TYPE));
    if (hipGetLastError() != hipSuccess) {
        cout << "cudaMallocManged failed" << endl;
    hipMallocManaged(&hz_gpu, NX * NY * sizeof(DATA_TYPE));
    if (hipGetLastError() != hipSuccess) {
        cout << "cudaMallocManged failed" << endl;
    }
    srand(1);
    init_arrays(_fict_, ex, ey, hz_cpu);

    int pass = 0;
    for (; pass < passes; pass++) {
        run_fdtd_cuda(_fict_, ex, ey, hz_cpu, hz_gpu);
    }
#endif
    

    
}
