#include "OptionParser.h"
#include "ResultDatabase.h"
#include "cudacommon.h"
#include <stdio.h>

// ****************************************************************************
// Function: addBenchmarkSpecOptions
//
// Purpose:
//   Add benchmark specific command line argument parsing.
//
//   -nopinned
//   This option controls whether page-locked or "pinned" memory is used.
//   The use of pinned memory typically results in higher bandwidth for data
//   transfer between host and device.
//
// Arguments:
//   op: the options parser / parameter database
//
// Returns:  nothing
//
// Programmer: Jeremy Meredith
// Creation: September 08, 2009
//
// Modifications:
//
// ****************************************************************************
void addBenchmarkSpecOptions(OptionParser &op) {
  op.addOption("pinned", OPT_BOOL, "1", "use pinned (pagelocked) memory");
}

// ****************************************************************************
// Function: runBenchmark
//
// Purpose:
//   Measures the bandwidth of the bus connecting the host processor to the
//   OpenCL device.  This benchmark repeatedly transfers data chunks of various
//   sizes across the bus to the OpenCL device, and calculates the bandwidth.
//
//
// Arguments:
//  resultDB: the benchmark stores its results in this ResultDatabase
//  op: the options parser / parameter database
//
// Returns:  nothing
//
// Programmer: Jeremy Meredith
// Creation: September 08, 2009
//
// Modifications:
//    Jeremy Meredith, Wed Dec  1 17:05:27 EST 2010
//    Added calculation of latency estimate.
//
// ****************************************************************************
void RunBenchmark(ResultDatabase &resultDB, OptionParser &op) {
  cout << "Running BusSpeedDownload" << endl;
  const bool verbose = op.getOptionBool("verbose");
  const bool quiet = op.getOptionBool("quiet");
  const bool pinned = op.getOptionBool("pinned");
  const int id = op.getOptionInt("device");

  // Sizes are in kb
  int nSizes = 20;
  int sizes[20] = {1,     2,     4,     8,      16,     32,    64,
                   128,   256,   512,   1024,   2048,   4096,  8192,
                   16384, 32768, 65536, 131072, 262144, 524288};
  long long numMaxFloats = 1024 * (sizes[nSizes - 1]) / 4;

  // Create some host memory pattern
  float *hostMem = NULL;
  if (pinned) {
#ifdef UNIFIED_MEMORY
    CUDA_SAFE_CALL(hipMallocManaged((void **)&hostMem, sizeof(float) * numMaxFloats));
    CUDA_SAFE_CALL(hipMemPrefetchAsync(hostMem, sizeof(float) * numMaxFloats, hipCpuDeviceId));
#else
    hipHostMalloc((void **)&hostMem, sizeof(float) * numMaxFloats);
#endif
    while (hipGetLastError() != hipSuccess) {
      // drop the size and try again
      if (verbose && !quiet) {
        cout << " - dropping size allocating pinned mem\n";
      }
      --nSizes;
      if (nSizes < 1) {
        cerr << "Error: Couldn't allocated any pinned buffer\n";
        return;
      }
      numMaxFloats = 1024 * (sizes[nSizes - 1]) / 4;
#ifdef UNIFIED_MEMORY
    CUDA_SAFE_CALL(hipMallocManaged((void **)&hostMem, sizeof(float) * numMaxFloats));
    CUDA_SAFE_CALL(hipMemPrefetchAsync(hostMem, sizeof(float) * numMaxFloats, hipCpuDeviceId));
#else
      hipHostMalloc((void **)&hostMem, sizeof(float) * numMaxFloats);
#endif
    }
  } else {
    hostMem = new float[numMaxFloats];
  }

  for (int i = 0; i < numMaxFloats; i++) {
    hostMem[i] = i % 77;
  }

  float *device;
#ifdef UNIFIED_MEMORY
  if (pinned)
    device = hostMem;
  else {
    hipMallocManaged((void **)&device, sizeof(float) * numMaxFloats);
    CUDA_SAFE_CALL(hipMemPrefetchAsync(device, sizeof(float) * numMaxFloats, id));
  }
#else
  hipMalloc((void **)&device, sizeof(float) * numMaxFloats);
#endif
  while (hipGetLastError() != hipSuccess) {
    // drop the size and try again
    if (verbose && !quiet) {
      cout << " - dropping size allocating device mem\n";
    }
    --nSizes;
    if (nSizes < 1) {
      cerr << "Error: Couldn't allocated any device buffer\n";
      return;
    }
    numMaxFloats = 1024 * (sizes[nSizes - 1]) / 4;
#ifdef UNIFIED_MEMORY
    if (pinned)
        device = hostMem;
    else {
        hipMallocManaged((void **)&device, sizeof(float) * numMaxFloats);
        CUDA_SAFE_CALL(hipMemPrefetchAsync(device, sizeof(float) * numMaxFloats, id));
    }
#else
    hipMalloc((void **)&device, sizeof(float) * numMaxFloats);
#endif
  }

  const unsigned int passes = op.getOptionInt("passes");

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Three passes, forward and backward both
  for (int pass = 0; pass < passes; pass++) {
    // store the times temporarily to estimate latency
    // float times[nSizes];
    // Step through sizes forward on even passes and backward on odd
    for (int i = 0; i < nSizes; i++) {
      int sizeIndex;
      if ((pass % 2) == 0)
        sizeIndex = i;
      else
        sizeIndex = (nSizes - 1) - i;

      int nbytes = sizes[sizeIndex] * 1024;

      hipEventRecord(start, 0);
#ifdef UNIFIED_MEMORY
      if (pinned)
        CUDA_SAFE_CALL(hipMemPrefetchAsync(device, nbytes, id));
      else
        CUDA_SAFE_CALL(hipMemcpy(device, hostMem, nbytes, hipMemcpyHostToDevice));

      hipDeviceSynchronize();

      // wait 
#else
      CUDA_SAFE_CALL(hipMemcpy(device, hostMem, nbytes, hipMemcpyHostToDevice));
#endif
      hipEventRecord(stop, 0);
      hipEventSynchronize(stop);
      float t = 0;
      hipEventElapsedTime(&t, start, stop);
      // times[sizeIndex] = t;

      // Convert to GB/sec
      if (verbose && !quiet) {
        cout << "size " << sizes[sizeIndex] << "k took " << t << " ms\n";
      }

      double speed = (double(sizes[sizeIndex]) * 1024. / (1000 * 1000)) / t;
      resultDB.AddResult("DownloadSpeed", "---", "GB/sec", speed);
      resultDB.AddOverall("DownloadSpeed", "GB/sec", speed);
    }
  }

  // Cleanup
  CUDA_SAFE_CALL(hipFree((void *)device));
  if (pinned) {
#ifndef UNIFIED_MEMORY
    CUDA_SAFE_CALL(hipHostFree((void *)hostMem));
#endif
  } else {
    delete[] hostMem;
  }
  CUDA_SAFE_CALL(hipEventDestroy(start));
  CUDA_SAFE_CALL(hipEventDestroy(stop));
}
