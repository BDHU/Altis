////////////////////////////////////////////////////////////////////////////////////////////////////
// file:	C:\Users\ed\source\repos\altis\src\cuda\level1\gemm\Gemm.cu
//
// summary:	Gemm class
// 
// origin: SHOC (https://github.com/vetter/shocp)
////////////////////////////////////////////////////////////////////////////////////////////////////

#include "OptionParser.h"
#include "ResultDatabase.h"
#include "Timer.h"
#include "Utility.h"
#include "hipblas.h"
//#include "hipblas.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "cudacommon.h"
#include "hip/hip_fp16.h"
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>

#define SEED 7
/// <summary>	Length of the object field. </summary>
static const int FIELD_LENGTH = 128;

using namespace std;

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Executes the test operation. </summary>
///
/// <typeparam name="T">	Generic type parameter. </typeparam>
/// <param name="testName">	Name of the test. </param>
/// <param name="resultDB">	[in,out] The result database. </param>
/// <param name="op">	   	[in,out] The operation. </param>
/// <param name="is_half"> 	Whether to use half precision. </param>
////////////////////////////////////////////////////////////////////////////////////////////////////

template <class T>
void RunTest(string testName, ResultDatabase &resultDB, OptionParser &op, bool is_half);

// origianlly don't need handle in v1 cublas

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Gemm operation wrapper. </summary>
///
/// <typeparam name="T">	Generic type parameter. </typeparam>
/// <param name="transa">	The transa. </param>
/// <param name="transb">	The transb. </param>
/// <param name="m">	 	An int to process. </param>
/// <param name="n">	 	An int to process. </param>
/// <param name="k">	 	An int to process. </param>
/// <param name="alpha"> 	The alpha. </param>
/// <param name="A">	 	A T to process. </param>
/// <param name="lda">   	The lda. </param>
/// <param name="B">	 	A T to process. </param>
/// <param name="ldb">   	The ldb. </param>
/// <param name="beta">  	The beta. </param>
/// <param name="C">	 	[in,out] If non-null, a T to process. </param>
/// <param name="ldc">   	The ldc. </param>
////////////////////////////////////////////////////////////////////////////////////////////////////

template <class T>
inline void devGEMM(char transa, char transb, int m, int n, int k, T alpha,
                    const T *A, int lda, const T *B, int ldb, T beta, T *C,
                    int ldc);


////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Filling memory. </summary>
///
/// <typeparam name="T">	Generic type parameter. </typeparam>
/// <param name="A">   	[in,out] If non-null,  pointer to the array to initialize. </param>
/// <param name="n">   number of elements in the array. </param>
/// <param name="maxi">	The maxi. </param>
////////////////////////////////////////////////////////////////////////////////////////////////////

template <class T> void fill(T *A, int n, int maxi) {
  for (int j = 0; j < n; j++) {
    A[j] = T((rand() % (maxi * 2 + 1)) - maxi) / (maxi + 1.);
  }
}


////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Reads a matrix. </summary>
///
/// <typeparam name="T">	Generic type parameter. </typeparam>
/// <param name="A">	   	[in,out] If non-null, pointer to matrix A. </param>
/// <param name="B">	   	[in,out] If non-null, pointer to matrix B. </param>
/// <param name="C">	   	[in,out] If non-null, pointer to matrix C. </param>
/// <param name="n">	   	An int to process. </param>
/// <param name="filename">	Filename of the file. </param>
////////////////////////////////////////////////////////////////////////////////////////////////////

template <class T> void readMatrix(T *A, T *B, T *C, int n, string filename) {
  std::ifstream mfs(filename.c_str());
  string line;
  // Ignore header line because it was already checked
  getline(mfs, line);
  float a, b, c;
  for (int j = 0; j < n; j++) {
    sscanf(line.c_str(), "%f %f %f", &a, &b, &c);
    A[j] = T(a);
    B[j] = T(b);
    C[j] = T(c);
  }
}

// ****************************************************************************
// Function: addBenchmarkSpecOptions
//
// Purpose:
//   Add benchmark specific options parsing.  The user is allowed to specify
//   the size of the input data in kiB.
//
// Arguments:
//   op: the options parser / parameter database
//
// Programmer: Anthony Danalis
// Creation: September 08, 2009
// Returns:  nothing
//
// ****************************************************************************
void addBenchmarkSpecOptions(OptionParser &op) {}

// ****************************************************************************
// Function: runBenchmark
//
// Purpose:
//   This benchmark measures the performance of the single precision general
//   matrix multiplication (SGEMM) operation in GFLOPS.  Data transfer time
//   over the PCIe bus is not included in this measurement.
//
// Arguments:
//  resultDB: the benchmark stores its results in this ResultDatabase
//  op: the options parser / parameter database
//
// Returns:  nothing
//
// Programmer: Anthony Danalis
// Creation: September 08, 2009
//
// Modifications:
//
// ****************************************************************************
void RunBenchmark(ResultDatabase &resultDB, OptionParser &op) {
   cout << "Running GEMM" << endl;
  int device;
  hipGetDevice(&device);
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, device);

  srand(SEED);

  bool quiet = op.getOptionBool("quiet");

  if(!quiet) {
    cout << "Running single precision test" << endl;
  }
  RunTest<float>("SGEMM", resultDB, op, false);


  // Test to see if this device supports double precision
  if ((deviceProp.major == 1 && deviceProp.minor >= 3) ||
      (deviceProp.major >= 2)) {
    if(!quiet) {
        cout << "Running double precision test" << endl;
    }
    RunTest<double>("DGEMM", resultDB, op, false);
  }

  /*
  if ((deviceProp.major >= 6)) {
    if (!quiet) {
        cout << "Running half preicsion test" << endl;
    }
    RunTest<half>("HGEMM", resultDB, op, true);
  }
  */
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	Executes the test operation. </summary>
///
/// <typeparam name="T">	Generic type parameter. </typeparam>
/// <param name="testName">	Name of the test. </param>
/// <param name="resultDB">	[in,out] The result database. </param>
/// <param name="op">	   	[in,out] The operation. </param>
/// <param name="is_half"> 	Whether to execute half precision op. </param>
////////////////////////////////////////////////////////////////////////////////////////////////////

template <class T>
void RunTest(string testName, ResultDatabase &resultDB, OptionParser &op, bool is_half) {
  int passes = op.getOptionInt("passes");
  int device = op.getOptionInt("device");
  int kib;

  // Use preset problem size or read data from input file
  string filename = op.getOptionString("inputFile");
  if (filename == "") {
    int probSizes[4] = {1, 3, 40, 60};
    kib = probSizes[op.getOptionInt("size") - 1];
  } else {
    std::ifstream mfs(filename.c_str());
    std::string line;
    char object[FIELD_LENGTH];
    sscanf(line.c_str(), "%s %d", object, &kib);
  }

  // Dimensions of matrix
  int N = kib * 1024 / sizeof(T);

  // Initialize the cublas library
  cublasInit();

  // Allocate GPU memory
#ifdef UNIFIED_MEMORY
  T *dA, *dB, *dC;
  CUDA_SAFE_CALL(hipMallocManaged(&dA, N * N* sizeof(T)));
  CUDA_SAFE_CALL(hipMallocManaged(&dB, N * N* sizeof(T)));
  CUDA_SAFE_CALL(hipMallocManaged(&dC, N * N* sizeof(T)));

  if (filename == "") {
    fill<T>(dA, N * N, 31);
    fill<T>(dB, N * N, 31);
    fill<T>(dC, N * N, 31);
  } else {
      readMatrix(dA, dB, dC, N * N, filename);
  }
#else
  T *dA, *dB, *dC;
  CUDA_SAFE_CALL(hipMalloc(&dA, N * N * sizeof(T)));
  CUDA_SAFE_CALL(hipMalloc(&dB, N * N * sizeof(T)));
  CUDA_SAFE_CALL(hipMalloc(&dC, N * N * sizeof(T)));

  // Initialize host memory
  T *A;
  T *B;
  T *C;

  CUDA_SAFE_CALL(hipHostMalloc(&A, N * N * sizeof(T)));
  CUDA_SAFE_CALL(hipHostMalloc(&B, N * N * sizeof(T)));
  CUDA_SAFE_CALL(hipHostMalloc(&C, N * N * sizeof(T)));

  // Fill matrix or read from input file
  if (filename == "") {
    fill<T>(A, N * N, 31);
    fill<T>(B, N * N, 31);
    fill<T>(C, N * N, 31);
  } else {
    readMatrix(A, B, C, N * N, filename);
  }
#endif

  // Copy input to GPU
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float elapsedTime;

  // Copy inputs to GPU

  double transferTime = 0;
  hipEventRecord(start, 0);
#ifdef UNIFIED_MEMORY
  // could ignore this to test demand paging performance affect
  CUDA_SAFE_CALL(hipMemPrefetchAsync(dA, N * N * sizeof(T), device));
  CUDA_SAFE_CALL(hipMemPrefetchAsync(dB, N * N * sizeof(T), device));
#else
  CUDA_SAFE_CALL(hipMemcpy(dA, A, N * N * sizeof(T), hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(dB, B, N * N * sizeof(T), hipMemcpyHostToDevice));
#endif

  hipblasHandle_t handle; // CUBLAS context
  //CUDA_SAFE_CALL(hipblasCreate(&handle));

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  transferTime += elapsedTime * 1.e-3;

  bool first = true;
/// <summary>	. </summary>
  for (int j = 0; j < passes; j++) {
    for (int i = 0; i < 2; i++) {
      const char transa = 'N';
      const char transb = i ? 'T' : 'N';
      const int nb = 128;
      const int idim = N / nb;

      int dim = idim * nb;

      const int m = dim;
      const int n = dim;
      const int k = dim;
      const int lda = dim;
      const int ldb = dim;
      const int ldc = dim;
      const T alpha = 1;
      const T beta = 0; //-1;

      // Warm Up
      devGEMM<T>(transa, transb, m, n, k, alpha, dA, lda, dB, ldb, beta, dC,
                    ldc);
      hipDeviceSynchronize();
      CHECK_CUDA_ERROR();

      double cublasTime;
      float kernelTime = 0.0f;
      for (int ii = 0; ii < 4; ++ii) {
        hipEventRecord(start, 0);
        devGEMM<T>(transa, transb, m, n, k, alpha, dA, lda, dB, ldb, beta, dC,
                   ldc);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        CHECK_CUDA_ERROR();
        float currTime = 0.0f;
        hipEventElapsedTime(&currTime, start, stop);
        kernelTime += currTime;
      }
      cublasTime = (kernelTime / 4.0) * 1.e-3;

      hipEventRecord(start, 0);    // timing may be affected by async
#ifdef UNIFIED_MEMORY
      CUDA_SAFE_CALL(hipMemPrefetchAsync(dC, N * N * sizeof(float), hipCpuDeviceId));
#else
      CUDA_SAFE_CALL(
          hipMemcpy(C, dC, N * N * sizeof(float), hipMemcpyDeviceToHost));
#endif
      hipEventRecord(stop, 0);
      hipEventSynchronize(stop);
      float oTransferTime = 0.0f;
      hipEventElapsedTime(&oTransferTime, start, stop);
      oTransferTime *= 1.e-3;

      // Add the PCIe transfer time to total transfer time only once
      if (first) {
        transferTime += oTransferTime;
        first = false;
      }

      double cublasGflops = 2. * m * n * k / cublasTime / 1e9;
      double pcieGflops = 2. * m * n * k / (cublasTime + transferTime) / 1e9;
      string atts = "dim:" + toString(dim);
      resultDB.AddResult(testName + "-" + transb + "-TransferTime", atts, "sec", transferTime);
      resultDB.AddResult(testName + "-" + transb + "-KernelTime", atts, "sec", cublasTime);
      resultDB.AddResult(testName + "-" + transb + "-TotalTime", atts, "sec", transferTime + cublasTime);
      resultDB.AddResult(testName + "-" + transb, atts, "GFlops", cublasGflops);
      resultDB.AddResult(testName + "-" + transb + "_PCIe", atts, "GFlops", pcieGflops);
      resultDB.AddResult(testName + "-" + transb + "_Parity", atts, "N", transferTime / cublasTime);
      resultDB.AddOverall("GFlops", "", cublasGflops);
    }
  }

  // Clean Up

  CUDA_SAFE_CALL(hipFree(dA));

  CUDA_SAFE_CALL(hipFree(dB));

  CUDA_SAFE_CALL(hipFree(dC));
#ifndef UNIFIED_MEMORY

  CUDA_SAFE_CALL(hipHostFree(A));

  CUDA_SAFE_CALL(hipHostFree(B));

  CUDA_SAFE_CALL(hipHostFree(C));
#endif

  CUDA_SAFE_CALL(hipEventDestroy(start));

  CUDA_SAFE_CALL(hipEventDestroy(stop));
  //hipblasDestroy(handle);

  cublasShutdown();
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>   gemm kernel (double). </summary>
///
/// <typeparam name="double">	Type of the double. </typeparam>
/// <param name="transa">	The transa. </param>
/// <param name="transb">	The transb. </param>
/// <param name="m">	 	An int to process. </param>
/// <param name="n">	 	An int to process. </param>
/// <param name="k">	 	An int to process. </param>
/// <param name="alpha"> 	The alpha. </param>
/// <param name="A">	 	A double to process. </param>
/// <param name="lda">   	The lda. </param>
/// <param name="B">	 	A double to process. </param>
/// <param name="ldb">   	The ldb. </param>
/// <param name="beta">  	The beta. </param>
/// <param name="C">	 	[in,out] If non-null, a double to process. </param>
/// <param name="ldc">   	The ldc. </param>
////////////////////////////////////////////////////////////////////////////////////////////////////

template <>
inline void devGEMM<double>(char transa, char transb, int m, int n, int k,
                            double alpha, const double *A, int lda,
                            const double *B, int ldb, double beta, double *C,
                            int ldc) {
  hipblasDgemm(transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}

////////////////////////////////////////////////////////////////////////////////////////////////////
/// <summary>	gemm kernel (float). </summary>
///
/// <typeparam name="float">	Type of the float. </typeparam>
/// <param name="transa">	The transa. </param>
/// <param name="transb">	The transb. </param>
/// <param name="m">	 	An int to process. </param>
/// <param name="n">	 	An int to process. </param>
/// <param name="k">	 	An int to process. </param>
/// <param name="alpha"> 	The alpha. </param>
/// <param name="A">	 	A float to process. </param>
/// <param name="lda">   	The lda. </param>
/// <param name="B">	 	A float to process. </param>
/// <param name="ldb">   	The ldb. </param>
/// <param name="beta">  	The beta. </param>
/// <param name="C">	 	[in,out] If non-null, a float to process. </param>
/// <param name="ldc">   	The ldc. </param>
////////////////////////////////////////////////////////////////////////////////////////////////////

template <>
inline void devGEMM<float>(char transa, char transb, int m, int n, int k,
                           float alpha, const float *A, int lda, const float *B,
                           int ldb, float beta, float *C, int ldc) {
  hipblasSgemm(transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}

/*
template <>
inline void devGEMM<half>(hipblasHandle_t handle, char transa, char transb, int m, int n, int k,
                            half alpha, const half *A, int lda,
                            const half *B, int ldb, half beta, half *C,
                            int ldc) {
  hipblasHgemm(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}
*/

